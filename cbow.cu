#include "hip/hip_runtime.h"
__device__ real reduceInWarp(real f, int idInWarp){
	for(int i = warpSize/2; i > 0; i /= 2){
		f += __shfl_xor(f, i, 32);
	}
	return f;
}

template<long long hs>
void __global__ cbow_kernel(long window, long negative, float alpha, long sentence_length, const int* __restrict__ sen, long layer1_size, volatile float *syn0, volatile float *syn1, const float* __restrict__ expTable, const int* __restrict__ vocab_codelen, const char* __restrict__ vocab_code, const int* __restrict__ vocab_point, const int* __restrict__ table, long table_size, long vocab_size, volatile float *syn1neg){
	extern __shared__ real s[]; //2*(real *)calloc(layer1_size, sizeof(real));

	int numWarpsPerBlock = blockDim.x/warpSize;
	int warpIdInBlock = threadIdx.x / warpSize;
	int warpId = warpIdInBlock + numWarpsPerBlock*blockIdx.x;
	int idInWarp = threadIdx.x % warpSize;

	real *neu1 = s + warpIdInBlock * layer1_size;
	real *neu1e = s + (numWarpsPerBlock+warpIdInBlock) * layer1_size;
	volatile int* cw = (volatile int*)(s + 2*numWarpsPerBlock*layer1_size);

	volatile unsigned long *temp_rand = (volatile unsigned long*)(s+2*numWarpsPerBlock*layer1_size) + numWarpsPerBlock;

	for(int sentence_position = warpId; sentence_position < sentence_length; sentence_position += gridDim.x*numWarpsPerBlock){
		long long word = sen[sentence_position];
		if (word == -1) continue;
		if(0 == idInWarp) {
			temp_rand[warpIdInBlock] = sentence_position;
			temp_rand[warpIdInBlock] = temp_rand[warpIdInBlock] * (unsigned long)25214903917 + 11;
		}
		unsigned long next_random = temp_rand[warpIdInBlock];//rand();
		int b = next_random % window;
		for (int c = idInWarp; c < layer1_size; c += warpSize) neu1[c] = 0;
		for (int c = idInWarp; c < layer1_size; c += warpSize) neu1e[c] = 0;
		// in -> hidden
		cw[warpIdInBlock] = 0;
		__syncthreads();

		for(int a = b; a < window * 2 + 1 - b; a++) if (a != window) {
			int c = sentence_position - window + a;
			if (c < 0) continue;
			if (c >= sentence_length) continue;
			long long last_word = sen[c];
			if (last_word == -1) continue;
			for (int c = idInWarp; c < layer1_size; c += warpSize) neu1[c] += syn0[c + last_word * layer1_size];
			if(idInWarp == 0) cw[warpIdInBlock]++;
		}
		if (cw[warpIdInBlock]) {
			for (int c = idInWarp; c < layer1_size; c += warpSize) neu1[c] /= cw[warpIdInBlock];
			if (hs) for (int d = vocab_codelen[word]; d < vocab_codelen[word+1]; d++) {
				float f = 0;
				int l2 = vocab_point[d] * layer1_size;
				// Propagate hidden -> output
				for (int c = idInWarp; c < layer1_size; c += warpSize) f += neu1[c] * syn1[c + l2];
				f = reduceInWarp(f, idInWarp);
				if (f <= -MAX_EXP) continue;
				else if (f >= MAX_EXP) continue;
				else f = expTable[(int)((f + MAX_EXP) * (EXP_TABLE_SIZE / MAX_EXP / 2))];
				// 'g' is the gradient multiplied by the learning rate
				float g = (1 - vocab_code[d] - f) * alpha;
				// Propagate errors output -> hidden
				for (int c = idInWarp; c < layer1_size; c += warpSize) neu1e[c] += g * syn1[c + l2];
				// Learn weights hidden -> output
				for (int c = idInWarp; c < layer1_size; c += warpSize) syn1[c + l2] += g * neu1[c];
			}
			// NEGATIVE SAMPLING
			if (negative > 0) for (int d = 0; d < negative + 1; d++) {
				int target;
				int label;
				if (d == 0) {
					target = word;
					label = 1;
				} else {
					if(0 == idInWarp) {
						temp_rand[warpIdInBlock] = temp_rand[warpIdInBlock] * (unsigned long)25214903917 + 11;
					}
					next_random = temp_rand[warpIdInBlock];
					//			
					target = table[(next_random >> 16) % table_size];
					if (target == 0) target = next_random % (vocab_size - 1) + 1;
					if (target == word) continue;
					label = 0;
				}
				int l2 = target * layer1_size;
				float f = 0;
				float g;
				for (int c = idInWarp; c < layer1_size; c += warpSize) f += neu1[c] * syn1neg[c + l2];
				f = reduceInWarp(f, idInWarp);
				if (f > MAX_EXP) g = (label - 1) * alpha;
				else if (f < -MAX_EXP) g = (label - 0) * alpha;
				else g = (label - expTable[(int)((f + MAX_EXP) * (EXP_TABLE_SIZE / MAX_EXP / 2))]) * alpha;
				for (int c = idInWarp; c < layer1_size; c += warpSize) neu1e[c] += g * syn1neg[c + l2];
				for (int c = idInWarp; c < layer1_size; c += warpSize) syn1neg[c + l2] += g * neu1[c];
			}
			// hidden -> in
			for (int a = b; a < window * 2 + 1 - b; a++) if (a != window) {
				int c = sentence_position - window + a;
				if (c < 0) continue;
				if (c >= sentence_length) continue;
				long long last_word = sen[c];
				if (last_word == -1) continue;
				for (int c = idInWarp; c < layer1_size; c += warpSize) syn0[c + last_word * layer1_size] += neu1e[c];
			}
		}
	}
}

void cbow_cuda(long window, long negative, float alpha, long sentence_length, int *sen, long layer1_size, float *syn0, long hs, float *syn1, float *expTable, int *vocab_codelen, char *vocab_code, int *vocab_point, int *table, long table_size, long vocab_size, float *syn1neg){
	int blockSize = 256;
	int gridSize = (sentence_length)/(blockSize/32);
	size_t smsize = (blockSize/32)*(2*layer1_size+3)*sizeof(float);
//printf("sm size is %d\n", smsize);
//fflush(stdout);
	cbow_kernel<1><<<gridSize, blockSize, smsize>>>(window, negative, alpha, sentence_length, sen, layer1_size, syn0, syn1, expTable, vocab_codelen, vocab_code, vocab_point, table, table_size, vocab_size, syn1neg);
}
